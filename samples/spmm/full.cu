/* Sample code for Sparse-Matrix-Dense Matrix multiplication.*/
#include <iostream>
#include <cstdlib>
#include <time.h>
#include <hip/hip_runtime.h>

#include <minigun/minigun.h>
#include "../samples_utils.h"
#include "../samples_io.h"

struct GData {
  int32_t dim = 0;
  float* cur{nullptr};
  float* next{nullptr};
  float* weight{nullptr};
  int* eid_mapping{nullptr};
};

struct SPMMFunctor {
  static __device__ __forceinline__ bool CondEdge(
      int32_t src, int32_t dst, int32_t eid, GData* gdata) {
    return true;
  }
  static __device__ __forceinline__ void ApplyEdge(
      int32_t src, int32_t dst, int32_t eid, GData* gdata) {}
  static __device__ __forceinline__ void ApplyEdgeReduce(
      int32_t src, int32_t dst, int32_t eid, int32_t feat_idx, float* val, GData* gdata) {
    *val += gdata->cur[src * gdata->dim + feat_idx] * gdata->weight[gdata->eid_mapping[eid]];
  }
  static __device__ __forceinline__ int32_t GetFeatSize(GData* gdata) {
    return gdata->dim;
  }
  static __device__ __forceinline__ float* GetOutBuf(GData* gdata) {
    return gdata->next;
  }
  static __device__ __forceinline__ int32_t GetOutOffset(int32_t idx, GData* gdata) {
    return idx;
  }
};

const int32_t D = 128;  // number of features

std::vector<float> GroundTruth(
    const std::vector<int32_t>& row_offsets,
    const std::vector<int32_t>& column_indices,
    const std::vector<float>& vdata,
    const std::vector<float>& edata) {
  std::vector<float> ret(vdata.size(), 0);
  for (size_t u = 0; u < row_offsets.size() - 1; ++u) {
    for (int32_t eid = row_offsets[u]; eid < row_offsets[u+1]; ++eid) {
      int32_t v = column_indices[eid];
      for (int32_t idx = 0; idx < D; ++idx) {
        ret[v * D + idx] += vdata[u * D + idx] * edata[eid];
      }
    }
  }
  return ret;
}

int main(int argc, char** argv) {
  srand(42);

  // create graph
  std::vector<int32_t> row_offsets, column_indices;
  utils::CreateNPGraph(1000, 0.01, row_offsets, column_indices);
  const int32_t N = row_offsets.size() - 1;
  const int32_t M = column_indices.size();
  std::cout << "#nodes: " << N << " #edges: " << M
    << " #feats: " << D << std::endl;

  // copy graph to gpu
  CUDA_CALL(hipSetDevice(0));
  minigun::IntCsr csr;
  minigun::IntArray infront;
  csr.row_offsets.length = row_offsets.size();
  CUDA_CALL(hipMalloc(&csr.row_offsets.data, sizeof(int32_t) * row_offsets.size()));
  CUDA_CALL(hipMemcpy(csr.row_offsets.data, &row_offsets[0],
        sizeof(int32_t) * row_offsets.size(), hipMemcpyHostToDevice));
  csr.column_indices.length = column_indices.size();
  CUDA_CALL(hipMalloc(&csr.column_indices.data, sizeof(int32_t) * column_indices.size()));
  CUDA_CALL(hipMemcpy(csr.column_indices.data, &column_indices[0],
        sizeof(int32_t) * column_indices.size(), hipMemcpyHostToDevice));
  csr.num_rows = N;
  csr.num_cols = N;

  // Create raw eid_mapping
  minigun::IntArray csr_mapping = utils::arange(0, M, kDLGPU);

  // Create csr_t and coo
  minigun::IntCsr csr_t;
  auto pack = utils::ToReverseCsr(csr, csr_mapping, kDLGPU);
  csr_t = pack.first;
  minigun::IntArray csr_t_mapping = pack.second;
  minigun::IntCoo coo;
  coo = utils::ToCoo(csr, kDLGPU);
  minigun::IntSpMat spmat = {&csr, &csr_t, &coo};

  // Create stream
  minigun::advance::RuntimeConfig config;
  config.ctx = {kDLGPU, 0};
  int nt = 1; //utils::_FindNumThreads(D, 32);
  config.data_num_threads = nt;
  config.data_num_blocks = 1;
  CUDA_CALL(hipStreamCreate(&config.stream));

  // Create feature data
  std::vector<float> vvec(N * D), evec(M);
  for (int32_t i = 0; i < N * D; ++i) {
    vvec[i] = (float)rand() / RAND_MAX;
  }
  for (int32_t i = 0; i < M; ++i) {
    evec[i] = (float)rand() / RAND_MAX;
  }

  // Copy feature data to gpu
  GData gdata;
  gdata.dim = D;
  CUDA_CALL(hipMalloc(&gdata.cur, sizeof(float) * N * D));
  CUDA_CALL(hipMemcpy(gdata.cur, &vvec[0], sizeof(float) * N * D, hipMemcpyHostToDevice));
  CUDA_CALL(hipMalloc(&gdata.next, sizeof(float) * N * D));
  CUDA_CALL(hipMemset(gdata.next, 0, sizeof(float) * N * D));
  CUDA_CALL(hipMalloc(&gdata.weight, sizeof(float) * M));
  CUDA_CALL(hipMemcpy(gdata.weight, &evec[0], sizeof(float) * M, hipMemcpyHostToDevice));
  gdata.eid_mapping = csr_t_mapping.data;

  CUDA_CALL(hipDeviceSynchronize());

  // Compute ground truth
  std::vector<float> truth = GroundTruth(row_offsets, column_indices,
      vvec, evec);

  typedef minigun::advance::Config<true, minigun::advance::kV2N, minigun::advance::kDst> Config;
  minigun::advance::Advance<kDLGPU, int32_t, float, Config, GData, SPMMFunctor>(
      config, spmat, &gdata, infront, nullptr);

  CUDA_CALL(hipDeviceSynchronize());

  // verify output
  std::vector<float> rst(N * D);
  CUDA_CALL(hipMemcpy(&rst[0], gdata.next, sizeof(float) * N * D, hipMemcpyDeviceToHost));

  std::cout << "Correct? " << utils::VecEqual(truth, rst) << std::endl;

  // free

  return 0;
}
