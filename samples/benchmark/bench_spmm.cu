#include <iostream>
#include <cstdlib>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#include <minigun/minigun.h>
#include "./baseline/yzh_kernels.cuh"
#include "./minigun/spmm.cuh"
#include "../samples_io.h"
#include "../samples_utils.h"

using minigun::advance::RuntimeConfig;
using namespace spmm;

double RunMinigun(const utils::SampleCsr& scsr,
                  const minigun::IntSpMat& spmat,
                  int32_t feat_size,
                  GData& gdata,
                  GData& truth) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // create stream
  RuntimeConfig rtcfg;
  rtcfg.ctx = {kDLGPU, 0};
  int nt = utils::_FindNumThreads(gdata.D, 512);
  rtcfg.data_num_threads = nt;
  rtcfg.data_num_blocks = (gdata.D + (nt * 4) - 1) / (nt * 4);
  CUDA_CALL(hipStreamCreate(&rtcfg.stream));

  ResetGData(&gdata, scsr.row_offsets.size() - 1);

  // check accuracy
  typedef minigun::advance::Config<minigun::advance::kDst> Config;
  minigun::advance::Advance<kDLGPU, int32_t, float, Config, GData, SPMMFunctor>(
      rtcfg, spmat, &gdata);
  CUDA_CALL(hipDeviceSynchronize());
  CheckResult(scsr, &gdata, &truth);

  // warm up
  const int K = 10;
  for (int i = 0; i < K; ++i) {
    minigun::advance::Advance<kDLGPU, int32_t, float, Config, GData, SPMMFunctor>(
        rtcfg, spmat, &gdata);
  }

  hipEventRecord(start);
  for (int i = 0; i < K; ++i) {
    minigun::advance::Advance<kDLGPU, int32_t, float, Config, GData, SPMMFunctor>(
        rtcfg, spmat, &gdata);
  }
  hipEventRecord(stop);
  CUDA_CALL(hipEventSynchronize(stop));
  float dur = 0;
  hipEventElapsedTime(&dur, start, stop);

  return dur / K;
}

double RunBaseline1(const utils::SampleCsr& scsr,
                    const minigun::IntCsr& csr,
                    int32_t feat_size,
                    GData& gdata,
                    GData& truth) {
  const int32_t N = csr.row_offsets.length - 1;
  ResetGData(&gdata, N);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int nt = utils::_FindNumThreads(gdata.D, 512);

  custom_kernel::vector_spmm_forward_kernel_no_eid<int32_t, float><<<N, nt>>>(
      csr.row_offsets.data,
      csr.column_indices.data,
      gdata.weight,
      gdata.ndata,
      gdata.out,
      (int)gdata.D, (int)N);
  CUDA_CALL(hipDeviceSynchronize());
  CheckResult(scsr, &gdata, &truth);

  const int K = 10;
  // warm up
  for (int i = 0; i < K; ++i) {
    custom_kernel::vector_spmm_forward_kernel_no_eid<int32_t, float><<<N, nt>>>(
        csr.row_offsets.data,
        csr.column_indices.data,
        gdata.weight,
        gdata.ndata,
        gdata.out,
        (int)gdata.D, (int)N);
  }

  hipEventRecord(start);
  for (int i = 0; i < K; ++i) {
    custom_kernel::vector_spmm_forward_kernel_no_eid<int32_t, float><<<N, nt>>>(
        csr.row_offsets.data,
        csr.column_indices.data,
        gdata.weight,
        gdata.ndata,
        gdata.out,
        (int)gdata.D, (int)N);
  }
  hipEventRecord(stop);
  CUDA_CALL(hipEventSynchronize(stop));
  float dur = 0;
  hipEventElapsedTime(&dur, start, stop);


  return dur / K;
}

double RunBaseline2(const utils::SampleCsr& scsr,
                    const minigun::IntCsr& csr,
                    int32_t feat_size,
                    GData& gdata,
                    GData& truth) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int n = csr.row_offsets.length - 1;
  ResetGData(&gdata, n);
  int k = feat_size;
  int nnz = scsr.row_offsets[n];

  float alpha = 1.0;
  float beta = 0.0;

  hipblasStatus_t stat;
  hipblasHandle_t cublas_handle{nullptr};
  stat = hipblasCreate(&cublas_handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("CUBLAS initialization failed\n");
    return EXIT_FAILURE;
  }

  hipsparseStatus_t status;
  hipsparseHandle_t handle=0;
  hipsparseMatDescr_t descr=0;

  /* initialize cusparse library */
  status= hipsparseCreate(&handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    printf("CUSPARSE Library initialization failed\n");
    exit(-1);
  }

  /* create and setup matrix descriptor */
  status= hipsparseCreateMatDescr(&descr);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    printf("Matrix descriptor initialization failed\n");
    exit(-1);
  }
  hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

  status= hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
          n, k, n, nnz, &alpha, descr, gdata.weight, csr.row_offsets.data, csr.column_indices.data,
          gdata.ndata, k, &beta, gdata.out, n);
  // transpose results to check correctness
  CUDA_CALL(hipDeviceSynchronize());
  float* t;
  CUDA_CALL(hipMalloc(&t, sizeof(float) * n * k));
  stat = hipblasSgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, k, n, &alpha, gdata.out, n, &beta, NULL, k, t, k);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("CUBLAS transpose failed\n");
    return EXIT_FAILURE;
  }
  GData gdata2;
  gdata2.out = t;
  CheckResult(scsr, &gdata2, &truth);
  CUDA_CALL(hipFree(t));
  CUDA_CALL(hipDeviceSynchronize());

  const int K = 10;

  // warm up
  for (int i = 0; i < K; ++i) {
    status= hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
            n, k, n, nnz, &alpha, descr, gdata.weight, csr.row_offsets.data, csr.column_indices.data,
            gdata.ndata, k, &beta, gdata.out, n);
  }
  hipEventRecord(start);
  for (int i = 0; i < K; ++i) {
    status= hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
            n, k, n, nnz, &alpha, descr, gdata.weight, csr.row_offsets.data, csr.column_indices.data,
            gdata.ndata, k, &beta, gdata.out, n);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float dur = 0;
  hipEventElapsedTime(&dur, start, stop);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    printf("Matrix-vector multiplication failed\n");
    exit(-1);
  }

  /* destroy matrix descriptor */
  status = hipsparseDestroyMatDescr(descr);
  descr = 0;
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    printf("Matrix descriptor destruction failed\n");
    exit(-1);
  }

  /* destroy handle */
  status = hipsparseDestroy(handle);
  handle = 0;
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    printf("CUSPARSE Library release of resources failed\n");
    exit(-1);
  }
  hipsparseDestroyMatDescr(descr);
  hipsparseDestroy(handle);
  hipblasDestroy(cublas_handle);
  return dur / K;
}

int main(int argc, char** argv) {
  // test transpose
  srand(42);
  /*
  // Small testing graph
  const int32_t a[] = {0, 2, 5, 5, 7};
  const int32_t b[] = {1, 2, 0, 2, 3, 0, 1};
  auto scsr = utils::SampleCsr{std::vector<int32_t>(std::begin(a), std::end(a)), std::vector<int32_t>(std::begin(b), std::end(b))};
  const int feat_size = 2;
  */
  if (argc < 3) {
    std::cout << "USAGE: ./bench_spmm <file_name> <feat_size>" << std::endl;
    return 1;
  }
  const char* filename = argv[1];
  const int feat_size = std::atoi(argv[2]);
  //std::cout << "filename=" << filename << " feat_size=" << feat_size

  utils::SampleCsr scsr;
  utils::LoadGraphFromFile(filename, &scsr);
  const int32_t N = scsr.row_offsets.size() - 1;
  const int32_t M = scsr.column_indices.size();
  //std::cout << "#Nodes: " << N << " #Edges: " << M << std::endl;

  // csr
  minigun::IntCsr csr = utils::ToMinigunCsr(scsr, kDLGPU);
  auto csr_mapping = utils::arange(0, M, kDLGPU);
  auto pack = utils::ToMinigunReverseCsr(scsr, csr_mapping, kDLGPU);
  minigun::IntCsr csr_t = pack.first;
  minigun::IntArray csr_t_mapping = pack.second;
  minigun::IntSpMat spmat = {nullptr, &csr_t, nullptr};

  // gdata
  GData gdata, truth;
  gdata.D = feat_size;
  InitGData(scsr, csr_t_mapping, &gdata, &truth);
  CUDA_CALL(hipDeviceSynchronize());

  //double dur1 = 0;
  double dur1 = RunBaseline1(scsr, csr_t, feat_size, gdata, truth);
  //double dur2 = 0;
  double dur2 = RunBaseline2(scsr, csr_t, feat_size, gdata, truth);
  //double dur3 = 0;
  double dur3 = RunMinigun(scsr, spmat, feat_size, gdata, truth);
  std::cout << N << "," << M << "," << feat_size << "," << dur1 << "," << dur2 << "," << dur3 << "\n";
  FreeGData(&gdata, &truth);
  hipDeviceReset();
  return 0;
}
