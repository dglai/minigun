#include <iostream>
#include <cstdlib>
#include <limits>
#include <time.h>
#include <hip/hip_runtime.h>

#include <minigun/minigun.h>
#include "./baseline/yzh_kernels.cuh"
#include "./minigun/esoftmax.cuh"
#include "../samples_io.h"
#include "../samples_utils.h"

using minigun::advance::RuntimeConfig;
using namespace esoftmax;

double RunMinigun(const utils::SampleCsr& scsr,
                  const minigun::IntCsr& csr,
                  int32_t feat_size, int32_t num_heads) {
  // gdata
  GData gdata, truth;
  gdata.H = num_heads;
  InitGData(scsr, &gdata, &truth);
  CUDA_CALL(hipDeviceSynchronize());
 
  // create stream
  RuntimeConfig rtcfg;
  rtcfg.ctx = {kDLGPU, 0};
  int nt = utils::_FindNumThreads(gdata.H, 32);
  rtcfg.data_num_threads = nt;
  rtcfg.data_num_blocks = gdata.H / nt;
  CUDA_CALL(hipStreamCreate(&rtcfg.stream));

  minigun::IntArray infront;

  // dry run
  typedef minigun::advance::Config<true, minigun::advance::kV2N> Config;
  minigun::advance::Advance<kDLGPU, int32_t, Config, GData, EdgeMax>(
      rtcfg, csr, &gdata, infront);
  minigun::advance::Advance<kDLGPU, int32_t, Config, GData, MinusMaxExpSum>(
      rtcfg, csr, &gdata, infront);
  minigun::advance::Advance<kDLGPU, int32_t, Config, GData, Norm>(
      rtcfg, csr, &gdata, infront);
  CUDA_CALL(hipDeviceSynchronize());
  CheckResult(scsr, &gdata, &truth);

  const int K = 10;
  timeval t0, t1;
  gettimeofday(&t0, nullptr);
  for (int i = 0; i < K; ++i) {
    minigun::advance::Advance<kDLGPU, int32_t, Config, GData, EdgeMax>(
        rtcfg, csr, &gdata, infront);
    minigun::advance::Advance<kDLGPU, int32_t, Config, GData, MinusMaxExpSum>(
        rtcfg, csr, &gdata, infront);
    minigun::advance::Advance<kDLGPU, int32_t, Config, GData, Norm>(
        rtcfg, csr, &gdata, infront);
  }
  CUDA_CALL(hipDeviceSynchronize());
  gettimeofday(&t1, nullptr);
  double dur = (double)(t1.tv_sec * 1e6 + t1.tv_usec -
      (t0.tv_sec * 1e6 + t0.tv_usec)) / K / 1000.0;  // ms

  FreeGData(&gdata, &truth);

  return dur;
}

double RunBaseline1(const utils::SampleCsr& scsr,
                  const minigun::IntCsr& csr,
                  int32_t feat_size, int32_t num_heads) {
  // gdata
  GData gdata, truth;
  gdata.H = num_heads;
  InitGData(scsr, &gdata, &truth);
 
  const int32_t N = csr.row_offsets.length - 1;
  const int H = gdata.H;

  // dry run
  custom_kernel::sparse_softmax_forward_kernel<int32_t, float><<<(N + 31) / 32, dim3(32, H)>>>(
      csr.row_offsets.data,
      gdata.score,
      gdata.ret,
      (int)N, (int)H);
  CUDA_CALL(hipDeviceSynchronize());

  const int K = 10;
  timeval t0, t1;
  gettimeofday(&t0, nullptr);
  for (int i = 0; i < K; ++i) {
    custom_kernel::sparse_softmax_forward_kernel<int32_t, float><<<(N + 31) / 32, dim3(32, H)>>>(
        csr.row_offsets.data,
        gdata.score,
        gdata.ret,
        (int)N, (int)H);
  }
  CUDA_CALL(hipDeviceSynchronize());
  gettimeofday(&t1, nullptr);
  double dur = (double)(t1.tv_sec * 1e6 + t1.tv_usec -
      (t0.tv_sec * 1e6 + t0.tv_usec)) / K / 1000.0;  // ms

  FreeGData(&gdata, &truth);

  return dur;
}

int main(int argc, char** argv) {
  srand(42);
  if (argc < 3) {
    std::cout << "USAGE: ./bench_masked_mm <file_name> <num_heads>" << std::endl;
    return 1;
  }
  const char* filename = argv[1];
  const int num_heads = std::atoi(argv[2]);
  std::cout << "filename=" << filename << " num_heads=" << num_heads << std::endl;

  utils::SampleCsr scsr;
  utils::LoadGraphFromFile(filename, &scsr);
  const int32_t N = scsr.row_offsets.size() - 1;
  const int32_t M = scsr.column_indices.size();
  std::cout << "#Nodes: " << N << " #Edges: " << M << std::endl;

  // csr
  minigun::IntCsr csr = utils::ToMinigunCsr(scsr, kDLGPU);

  double dur1 = RunMinigun(scsr, csr, 0, num_heads);
  std::cout << "minigun time(ms): " << dur1 << std::endl;
  double dur2 = RunBaseline1(scsr, csr, 0, num_heads);
  std::cout << "baseline1 time(ms): " << dur2 << std::endl;

  return 0;
}
