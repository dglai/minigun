#include "hip/hip_runtime.h"
/* Sample code for edge softmax.*/
#include <iostream>
#include <cstdlib>
#include <limits>
#include <time.h>
#include <hip/hip_runtime.h>

#include <minigun/minigun.h>
#include "../samples_utils.h"

struct GData {
  int32_t dim = 0;
  float* sum{nullptr};  // ndata
  float* max{nullptr};  // ndata
  float* score{nullptr};
};

__device__ __forceinline__ float MyAtomicMax(float* addr, float val) {
  uint32_t* addr_as_ui = reinterpret_cast<uint32_t*>(addr);
  uint32_t old = *addr_as_ui;
  uint32_t assumed = old;
  do {
    assumed = old;
    old = atomicCAS(addr_as_ui, assumed,
        __float_as_uint(fmax(val, __uint_as_float(old))));
  } while (assumed != old);
  return __uint_as_float(old);
}

// Max
struct EdgeMax {
  static __device__ __forceinline__ bool CondEdge(
      int32_t src, int32_t dst, int32_t eid, GData* gdata) {
    return true;
  }
  static __device__ __forceinline__ void ApplyEdge(
      int32_t src, int32_t dst, int32_t eid, GData* gdata) {
    int32_t tx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride_x = blockDim.x * gridDim.x;
    const int32_t dim = gdata->dim;
    while (tx < dim) {
      MyAtomicMax(gdata->max + dst * dim + tx, gdata->score[eid * dim + tx]);
      tx += stride_x;
    }
  }
};

// minus max, exp and sum
struct MinuxMaxExpSum {
  static __device__ __forceinline__ bool CondEdge(
      int32_t src, int32_t dst, int32_t eid, GData* gdata) {
    return true;
  }
  static __device__ __forceinline__ void ApplyEdge(
      int32_t src, int32_t dst, int32_t eid, GData* gdata) {
    int32_t tx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride_x = blockDim.x * gridDim.x;
    const int32_t dim = gdata->dim;
    while (tx < dim) {
      gdata->score[eid * dim + tx] = expf(
          gdata->score[eid * dim + tx] - gdata->max[dst * dim + tx]);
      atomicAdd(gdata->sum + dst * dim + tx, gdata->score[eid * dim + tx]);
      tx += stride_x;
    }
  }
};

// norm
struct Norm {
  static __device__ __forceinline__ bool CondEdge(
      int32_t src, int32_t dst, int32_t eid, GData* gdata) {
    return true;
  }
  static __device__ __forceinline__ void ApplyEdge(
      int32_t src, int32_t dst, int32_t eid, GData* gdata) {
    int32_t tx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride_x = blockDim.x * gridDim.x;
    const int32_t dim = gdata->dim;
    while (tx < dim) {
      gdata->score[eid * dim + tx] /= gdata->sum[dst * dim + tx];
      tx += stride_x;
    }
  }
};

const int32_t D = 8;  // number of heads

std::vector<float> GroundTruth(
    const std::vector<int32_t>& row_offsets,
    const std::vector<int32_t>& column_indices,
    std::vector<float> score) {
  const size_t N = row_offsets.size() - 1;
  std::vector<float> tmp(N * D, 0.);
  for (size_t i = 0; i < score.size(); ++i) {
    score[i] = std::exp(score[i]);
  }
  for (size_t u = 0; u < row_offsets.size() - 1; ++u) {
    for (int32_t eid = row_offsets[u]; eid < row_offsets[u+1]; ++eid) {
      int32_t v = column_indices[eid];
      for (int32_t idx = 0; idx < D; ++idx) {
        tmp[v * D + idx] += score[eid * D + idx];
      }
    }
  }
  for (size_t eid = 0; eid < column_indices.size(); ++eid) {
    for (int32_t i = 0; i < D; ++i) {
      score[eid * D + i] /= tmp[column_indices[eid] * D + i];
    }
  }
  return score;
}

int main(int argc, char** argv) {
  srand(42);

  // create graph
  std::vector<int32_t> row_offsets, column_indices;
  utils::CreateNPGraph(1000, 0.01, row_offsets, column_indices);
  const int32_t N = row_offsets.size() - 1;
  const int32_t M = column_indices.size();
  std::cout << "#nodes: " << N << " #edges: " << M
    << " #feats: " << D << std::endl;

  // copy graph to gpu
  CUDA_CALL(hipSetDevice(0));
  minigun::IntCsr csr;
  minigun::IntArray infront;
  csr.row_offsets.length = row_offsets.size();
  CUDA_CALL(hipMalloc(&csr.row_offsets.data, sizeof(int32_t) * row_offsets.size()));
  CUDA_CALL(hipMemcpy(csr.row_offsets.data, &row_offsets[0],
        sizeof(int32_t) * row_offsets.size(), hipMemcpyHostToDevice));
  csr.column_indices.length = column_indices.size();
  CUDA_CALL(hipMalloc(&csr.column_indices.data, sizeof(int32_t) * column_indices.size()));
  CUDA_CALL(hipMemcpy(csr.column_indices.data, &column_indices[0],
        sizeof(int32_t) * column_indices.size(), hipMemcpyHostToDevice));

  // Create stream
  minigun::advance::RuntimeConfig config;
  config.ctx = {kDLGPU, 0};
  int nt = utils::_FindNumThreads(D, 32);
  config.data_num_threads = nt;
  config.data_num_blocks = (M + nt - 1) / nt;
  CUDA_CALL(hipStreamCreate(&config.stream));

  // Create feature data
  std::vector<float> vvec(N * D), evec(M * D);
  for (int32_t i = 0; i < N * D; ++i) {
    vvec[i] = std::numeric_limits<float>::lowest();
  }
  for (int32_t i = 0; i < M * D; ++i) {
    evec[i] = (float)rand() / RAND_MAX - 0.5;
  }
  //utils::VecPrint(evec);

  // Copy feature data to gpu
  GData gdata;
  gdata.dim = D;
  CUDA_CALL(hipMalloc(&gdata.sum, sizeof(float) * N * D));
  CUDA_CALL(hipMemset(gdata.sum, 0, sizeof(float) * N * D));
  CUDA_CALL(hipMalloc(&gdata.max, sizeof(float) * N * D));
  CUDA_CALL(hipMemcpy(gdata.max, &vvec[0], sizeof(float) * N * D, hipMemcpyHostToDevice));
  CUDA_CALL(hipMalloc(&gdata.score, sizeof(float) * M * D));
  CUDA_CALL(hipMemcpy(gdata.score, &evec[0], sizeof(float) * M * D, hipMemcpyHostToDevice));

  CUDA_CALL(hipDeviceSynchronize());

  // Compute ground truth
  std::vector<float> truth = GroundTruth(row_offsets, column_indices, evec);
  //utils::VecPrint(truth);

  typedef minigun::advance::Config<true, minigun::advance::kV2N> Config;
  minigun::advance::Advance<kDLGPU, int32_t, Config, GData, EdgeMax>(
      config, csr, &gdata, infront);
  minigun::advance::Advance<kDLGPU, int32_t, Config, GData, MinuxMaxExpSum>(
      config, csr, &gdata, infront);
  minigun::advance::Advance<kDLGPU, int32_t, Config, GData, Norm>(
      config, csr, &gdata, infront);

  CUDA_CALL(hipDeviceSynchronize());

  // verify output
  std::vector<float> rst(M * D);
  CUDA_CALL(hipMemcpy(&rst[0], gdata.score, sizeof(float) * M * D, hipMemcpyDeviceToHost));
  //utils::VecPrint(rst);

  std::cout << "Correct? " << utils::VecEqual(truth, rst) << std::endl;

  // free

  return 0;
}
