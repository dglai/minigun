#include "hip/hip_runtime.h"
/* Sample code for edge softmax.*/
#include <iostream>
#include <cstdlib>
#include <limits>
#include <time.h>
#include <hip/hip_runtime.h>

#include <minigun/minigun.h>
#include "../samples_utils.h"

struct GData {
  mg_int dim = 0;
  float* sum{nullptr};  // ndata
  float* max{nullptr};  // ndata
  float* score{nullptr};
};

__device__ __forceinline__ float MyAtomicMax(float* addr, float val) {
  uint32_t* addr_as_ui = reinterpret_cast<uint32_t*>(addr);
  uint32_t old = *addr_as_ui;
  uint32_t assumed = old;
  do {
    assumed = old;
    old = atomicCAS(addr_as_ui, assumed,
        __float_as_uint(fmax(val, __uint_as_float(old))));
  } while (assumed != old);
  return __uint_as_float(old);
}

// Max
struct EdgeMax {
  static __device__ __forceinline__ bool CondEdge(
      mg_int src, mg_int dst, mg_int eid, GData* gdata) {
    return true;
  }
  static __device__ __forceinline__ void ApplyEdge(
      mg_int src, mg_int dst, mg_int eid, GData* gdata) {
    mg_int tx = blockIdx.x * blockDim.x + threadIdx.x;
    mg_int stride_x = blockDim.x * gridDim.x;
    const mg_int dim = gdata->dim;
    while (tx < dim) {
      MyAtomicMax(gdata->max + dst * dim + tx, gdata->score[eid * dim + tx]);
      tx += stride_x;
    }
  }
};

// minus max, exp and sum
struct MinuxMaxExpSum {
  static __device__ __forceinline__ bool CondEdge(
      mg_int src, mg_int dst, mg_int eid, GData* gdata) {
    return true;
  }
  static __device__ __forceinline__ void ApplyEdge(
      mg_int src, mg_int dst, mg_int eid, GData* gdata) {
    mg_int tx = blockIdx.x * blockDim.x + threadIdx.x;
    mg_int stride_x = blockDim.x * gridDim.x;
    const mg_int dim = gdata->dim;
    while (tx < dim) {
      gdata->score[eid * dim + tx] = expf(
          gdata->score[eid * dim + tx] - gdata->max[dst * dim + tx]);
      atomicAdd(gdata->sum + dst * dim + tx, gdata->score[eid * dim + tx]);
      tx += stride_x;
    }
  }
};

// norm
struct Norm {
  static __device__ __forceinline__ bool CondEdge(
      mg_int src, mg_int dst, mg_int eid, GData* gdata) {
    return true;
  }
  static __device__ __forceinline__ void ApplyEdge(
      mg_int src, mg_int dst, mg_int eid, GData* gdata) {
    mg_int tx = blockIdx.x * blockDim.x + threadIdx.x;
    mg_int stride_x = blockDim.x * gridDim.x;
    const mg_int dim = gdata->dim;
    while (tx < dim) {
      gdata->score[eid * dim + tx] /= gdata->sum[dst * dim + tx];
      tx += stride_x;
    }
  }
};

const mg_int D = 8;  // number of heads

std::vector<float> GroundTruth(
    const std::vector<mg_int>& row_offsets,
    const std::vector<mg_int>& column_indices,
    std::vector<float> score) {
  const size_t N = row_offsets.size() - 1;
  std::vector<float> tmp(N * D, 0.);
  for (size_t i = 0; i < score.size(); ++i) {
    score[i] = std::exp(score[i]);
  }
  for (size_t u = 0; u < row_offsets.size() - 1; ++u) {
    for (mg_int eid = row_offsets[u]; eid < row_offsets[u+1]; ++eid) {
      mg_int v = column_indices[eid];
      for (mg_int idx = 0; idx < D; ++idx) {
        tmp[v * D + idx] += score[eid * D + idx];
      }
    }
  }
  for (size_t eid = 0; eid < column_indices.size(); ++eid) {
    for (mg_int i = 0; i < D; ++i) {
      score[eid * D + i] /= tmp[column_indices[eid] * D + i];
    }
  }
  return score;
}

int main(int argc, char** argv) {
  srand(42);

  // create graph
  std::vector<mg_int> row_offsets, column_indices;
  utils::CreateNPGraph(1000, 0.01, row_offsets, column_indices);
  const mg_int N = row_offsets.size() - 1;
  const mg_int M = column_indices.size();
  std::cout << "#nodes: " << N << " #edges: " << M
    << " #feats: " << D << std::endl;

  // copy graph to gpu
  CUDA_CALL(hipSetDevice(0));
  minigun::Csr csr;
  minigun::IntArray1D infront;
  csr.row_offsets.length = row_offsets.size();
  CUDA_CALL(hipMalloc(&csr.row_offsets.data, sizeof(mg_int) * row_offsets.size()));
  CUDA_CALL(hipMemcpy(csr.row_offsets.data, &row_offsets[0],
        sizeof(mg_int) * row_offsets.size(), hipMemcpyHostToDevice));
  csr.column_indices.length = column_indices.size();
  CUDA_CALL(hipMalloc(&csr.column_indices.data, sizeof(mg_int) * column_indices.size()));
  CUDA_CALL(hipMemcpy(csr.column_indices.data, &column_indices[0],
        sizeof(mg_int) * column_indices.size(), hipMemcpyHostToDevice));

  // Create stream
  minigun::advance::RuntimeConfig config;
  config.ctx = {kDLGPU, 0};
  int nt = utils::_FindNumThreads(D, 32);
  config.data_num_threads = nt;
  config.data_num_blocks = (M + nt - 1) / nt;
  CUDA_CALL(hipStreamCreate(&config.stream));

  // Create feature data
  std::vector<float> vvec(N * D), evec(M * D);
  for (mg_int i = 0; i < N * D; ++i) {
    vvec[i] = std::numeric_limits<float>::lowest();
  }
  for (mg_int i = 0; i < M * D; ++i) {
    evec[i] = (float)rand() / RAND_MAX - 0.5;
  }
  //utils::VecPrint(evec);

  // Copy feature data to gpu
  GData gdata;
  gdata.dim = D;
  CUDA_CALL(hipMalloc(&gdata.sum, sizeof(float) * N * D));
  CUDA_CALL(hipMemset(gdata.sum, 0, sizeof(float) * N * D));
  CUDA_CALL(hipMalloc(&gdata.max, sizeof(float) * N * D));
  CUDA_CALL(hipMemcpy(gdata.max, &vvec[0], sizeof(float) * N * D, hipMemcpyHostToDevice));
  CUDA_CALL(hipMalloc(&gdata.score, sizeof(float) * M * D));
  CUDA_CALL(hipMemcpy(gdata.score, &evec[0], sizeof(float) * M * D, hipMemcpyHostToDevice));

  CUDA_CALL(hipDeviceSynchronize());

  // Compute ground truth
  std::vector<float> truth = GroundTruth(row_offsets, column_indices, evec);
  //utils::VecPrint(truth);

  typedef minigun::advance::Config<true, minigun::advance::kV2N> Config;
  minigun::advance::Advance<kDLGPU, Config, GData, EdgeMax>(
      config, csr, &gdata, infront);
  minigun::advance::Advance<kDLGPU, Config, GData, MinuxMaxExpSum>(
      config, csr, &gdata, infront);
  minigun::advance::Advance<kDLGPU, Config, GData, Norm>(
      config, csr, &gdata, infront);

  CUDA_CALL(hipDeviceSynchronize());

  // verify output
  std::vector<float> rst(M * D);
  CUDA_CALL(hipMemcpy(&rst[0], gdata.score, sizeof(float) * M * D, hipMemcpyDeviceToHost));
  //utils::VecPrint(rst);

  std::cout << "Correct? " << utils::VecEqual(truth, rst) << std::endl;

  // free

  return 0;
}
